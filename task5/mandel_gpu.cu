#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

/* Problem size */
#define XSIZE 2560
#define YSIZE 2048
/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 32
#define BLOCKX 32
#define MAXITER 255 // you may want to increase this

double xleft=-2.01;
double xright=1;
double yupper,ylower;
double ycenter=1e-6;
double step;

int host_pixel[XSIZE*YSIZE];
int device_pixel[XSIZE*YSIZE];

typedef struct {
    double real,imag;
} my_complex_t;

#define PIXEL(i,j) ((i)+(j)*XSIZE)

// ********** SUBTASK1: Create kernel device_calculate ******************/
//Insert code here
// Hint: Use _global_ for the kernal function to be executed on the GPU.
// Also set up a single grid with a 2D thread block

__global__ void device_calculate(int* device_pixel, double xleft, double yupper, double step) {
    // Calculate global thread indices
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    
    // Make sure i and j is within bounds.
    if (i < XSIZE && j < YSIZE) {
        my_complex_t c, z, temp;
        int iter = 0;
                
        // Map pixel coordinates to complex plane
        c.real = (xleft + step * i);
        c.imag = (yupper - step * j);
        z = c;
        
        // Iterate until escape or max iterations reached
        while (z.real * z.real + z.imag * z.imag < 4.0 && iter < MAXITER) {
            temp.real = z.real * z.real - z.imag * z.imag + c.real;
            temp.imag = 2.0 * z.real * z.imag + c.imag;
            z = temp;
            iter++;
        }
        // Store iteration count
        device_pixel[i + j * XSIZE] = iter;
    }
}
// ********** SUBTASK1 END ***********************************************/

void host_calculate() {
    for(int j = 0; j < YSIZE; j++) {
        for(int i = 0; i < XSIZE; i++) {
            /* Calculate the number of iterations until divergence for each pixel. If divergence never happens, return MAXITER */
            my_complex_t c, z, temp;
            int iter=0;
            c.real = (xleft + step*i);
            c.imag = (yupper - step*j);
            z = c;
            
            while(z.real*z.real + z.imag*z.imag<4.0) {
                temp.real = z.real*z.real - z.imag*z.imag + c.real;
                temp.imag = 2.0*z.real*z.imag + c.imag;
                z = temp;
                if(++iter==MAXITER) break;
            }
            host_pixel[PIXEL(i,j)]=iter;
        }
    }
}

typedef unsigned char uchar;

// save 24-bits bmp file, buffer must be in bmp format: upside-down
void savebmp(char *name, uchar *buffer, int x, int y) {
    FILE *f=fopen(name,"wb");

    if(!f) {
        printf("Error writing image to disk.\n");
        return;
    }

    unsigned int size=x*y*3+54;
    uchar header[54]={'B','M',size&255,(size>>8)&255,(size>>16)&255,size>>24,0,0,0,0,54,0,0,0,40,0,0,0,x&255,x>>8,0,0,y&255,y>>8,0,0,1,0,24,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    fwrite(header,1,54,f);
    fwrite(buffer,1,x*y*3,f);
    fclose(f);
}

// given iteration number, set a color
void fancycolour(uchar *p,int iter) {
    if(iter==MAXITER);
    else if(iter<8) { p[0]=128+iter*16; p[1]=p[2]=0; }
    else if(iter<24) { p[0]=255; p[1]=p[2]=(iter-8)*16; }
    else if(iter<160) { p[0]=p[1]=255-(iter-24)*2; p[2]=255; }
    else { p[0]=p[1]=(iter-160)*2; p[2]=255-(iter-160)*2; }
}

// Get system time to microsecond precision ostensibly, similar to MPI_Wtime), returns time in seconds
double walltime ( void ) {
    static struct timeval t;
    gettimeofday ( &t, NULL );
    return ( t.tv_sec + 1e-6 * t.tv_usec );
}

int main(int argc, char **argv) {
    if(argc==1) {
        puts("Usage: MANDEL n");
        puts("n decides whether image should be written to disk (1=yes, 0=no)");
        return 0;
    }
    double start;
    double hosttime=0;
    double devicetime=0;
    double memtime=0;

    hipDeviceProp_t p;
    hipSetDevice(0);
    hipGetDeviceProperties(&p, 0);
    printf("Device compute capability: %d.%d\n", p.major, p.minor);

    /* Calculate the range in the y-axis such that we preserve the aspect ratio */
    step=(xright-xleft)/XSIZE;
    yupper=ycenter+(step*YSIZE)/2;
    ylower=ycenter-(step*YSIZE)/2;

    /* Host calculates image */
    start=walltime();
    host_calculate();
    hosttime+=walltime()-start;

    //********** SUBTASK2: Set up device memory ***************************/
    // Define a variable for use in device calculations.
    int *d_pixel;
    // Allocates necessary size for d_pixel variable based on X/Y-SIZE.
    hipMalloc((void**)&d_pixel, XSIZE * YSIZE * sizeof(int));
    // Copies the memory address' content of device_pixel to d_pixel on the device (GPU)
    hipMemcpy(d_pixel, device_pixel, (XSIZE * YSIZE * sizeof(int)), hipMemcpyHostToDevice);
    /********** SUBTASK2 END **********************************************/

    start=walltime();

    //********* SUBTASK3: Execute the kernel on the device ************/
    // Define the dimensions of a thread block.
    dim3 block(BLOCKX, BLOCKY);
    // Calculate the dimensions of the grid of blocks. 
    // Even if XSIZE or YSIZE is not evenly divisible by BLOCKX or BLOCKY it will divide it as evenly as possible.
    dim3 grid((XSIZE + BLOCKX - 1) / BLOCKX, (YSIZE + BLOCKY - 1) / BLOCKY);
    // Launch the CUDA kernel
    device_calculate<<<grid, block>>>(d_pixel, xleft, yupper, step);
    // Synchronize the host (CPU) with the device (GPU) for timing accuracy and ensuring results are ready
    hipDeviceSynchronize();
    //********** SUBTASK3 END *****************************************/

    devicetime+=walltime()-start;

    start=walltime();

    //***** SUBTASK4: Transfer the result from device to device_pixel[][]*/
    // Copy the results from d_pixel gpu variable to the device_pixel for comparison and saving.
    hipMemcpy(device_pixel, d_pixel, (XSIZE * YSIZE * sizeof(int)), hipMemcpyDeviceToHost);
    //********** SUBTASK4 END ******************************************/

    memtime+=walltime()-start;

    /****** SUBTASK5: Free the device memory also ************************/
    // Free the d_pixel gpu variable.
    hipFree(d_pixel);
    /********** SUBTASK5 END ******************************************/

    int errors=0;
    /* check if result is correct */
    for(int i=0;i<XSIZE;i++) {
        for(int j=0;j<YSIZE;j++) {
            int diff=host_pixel[PIXEL(i,j)]-device_pixel[PIXEL(i,j)];
            if(diff<0) diff=-diff;
            /* allow +-1 difference */
            if(diff>1) {
                if(errors<10) printf("Error on pixel %d %d: expected %d, found %d\n", i,j,host_pixel[PIXEL(i,j)],device_pixel[PIXEL(i,j)]);
                else if(errors==10) puts("...");
                    errors++;
            }
        }
    }

    if(errors > 0) printf("Found %d errors.\n",errors);
    else puts("Device calculations are correct.");

    printf("\n");
    printf("Host time: %7.3f ms\n",hosttime*1e3);
    printf("Device calculation: %7.3f ms\n",devicetime*1e3);
    printf("Copy result: %7.3f ms\n",memtime*1e3);

    if(strtol(argv[1],NULL,10)!=0) {
        /* create nice image from iteration counts. take care to create it
        upside
        down (bmp format) */
        unsigned char *buffer=(unsigned char *)calloc(XSIZE*YSIZE*3,1);
        for(int i=0;i<XSIZE;i++) {
            for(int j=0;j<YSIZE;j++) {
                int p=((YSIZE-j-1)*XSIZE+i)*3;
                fancycolour(buffer+p,device_pixel[PIXEL(i,j)]);
            }
        }
        /* write image to disk */
        savebmp("mandel1.bmp",buffer,XSIZE,YSIZE);
    }
    return 0;
}