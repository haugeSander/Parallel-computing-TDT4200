#include "hip/hip_runtime.h"
//#define _XOPEN_SOURCE 600
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <sys/time.h>

// TASK: T1
// Include the cooperative groups library
// BEGIN: T1
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
// END: T1


// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

// Option to change numerical precision
typedef int64_t int_t;
typedef double real_t;

// TASK: T1b
// Variables needed for implementation
// BEGIN: T1b

// Simulation parameters: size, step count, and how often to save the state
int_t
    N = 128,
    M = 128,
    max_iteration = 100000,//0,
    snapshot_freq = 1000;

// Wave equation parameters, time step is derived from the space step
const real_t
    c  = 1.0,
    dx = 1.0,
    dy = 1.0;
real_t
    dt;

// Buffers for three time steps, indexed with 2 ghost points for the boundary
real_t
    *buffers[3] = { NULL, NULL, NULL };

#define U_prv(i,j) buffers[0][((i)+1)*(N+2)+(j)+1]
#define U(i,j)     buffers[1][((i)+1)*(N+2)+(j)+1]
#define U_nxt(i,j) buffers[2][((i)+1)*(N+2)+(j)+1]


// Divide the problem into blocks of BLOCKX x BLOCKY threads
#define BLOCKY 16
#define BLOCKX 16

// Global CUDA prop information
hipDeviceProp_t prop;

// Device-side variables
real_t *d_prv = NULL;   // Previous time step on device
real_t *d_current = NULL;  // Current time step on device
real_t *d_nxt = NULL;    // Next time step on device
// END: T1b

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Rotate the time step buffers.
void move_buffer_window ( void )
{
    real_t *temp = buffers[0];
    buffers[0] = buffers[1];
    buffers[1] = buffers[2];
    buffers[2] = temp;

    // Rotate pointers for next iteration
    real_t *d_temp = d_prv;
    d_prv = d_current;
    d_current = d_nxt;
    d_nxt = d_temp;
}

// Save the present time step in a numbered file under 'data/'
void domain_save ( int_t step )
{
    char filename[256];
    sprintf ( filename, "data/%.5ld.dat", step );
    FILE *out = fopen ( filename, "wb" );
    for ( int_t i=0; i<M; i++ )
    {
        fwrite ( &U(i,0), sizeof(real_t), N, out );
    }
    fclose ( out );
}


// TASK: T4
// Get rid of all the memory allocations
void domain_finalize ( void )
{
// BEGIN: T4
    // Free memory on host
    free ( buffers[0] );
    free ( buffers[1] );
    free ( buffers[2] );

    // Free memory on device
    hipFree(d_prv);
    hipFree(d_current);
    hipFree(d_nxt);
// END: T4
}


// TASK: T6
// Neumann (reflective) boundary condition
// BEGIN: T6
__global__ void device_boundary_condition(real_t* d_current, int N, int M) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Handle boundary conditions
    if (i < M) {
        if (j == 0) {
            d_current[(i+1)*(N+2)] = d_current[(i+1)*(N+2)+2];
        }
        if (j == N-1) {
            d_current[(i+1)*(N+2)+N+1] = d_current[(i+1)*(N+2)+N-1];
        }
    }
    if (j < N) {
        if (i == 0) {
            d_current[(j+1)] = d_current[2*(N+2)+(j+1)];
        }
        if (i == M-1) {
            d_current[(M+1)*(N+2)+(j+1)] = d_current[(M-1)*(N+2)+(j+1)];
        }
    }
}
// END: T6


// TASK: T5
// Integration formula
// BEGIN; T5
__global__ void device_time_step ( real_t* d_prv, real_t* d_current, real_t* d_nxt, int N, int M, real_t dt, real_t dx, real_t dy, real_t c )
{
    // Calculate global thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Synchronize the grid
    //cg::this_grid().sync(); 
    //device_boundary_condition(d_current, N, M);

    // Check if thread is within bounds
    if (i < M && j < N) {
        // Adjust indices for ghost cells
        int idx = (i+1)*(N+2) + (j+1);
        int idx_up = ((i-1)+1)*(N+2) + (j+1);
        int idx_down = ((i+1)+1)*(N+2) + (j+1);
        int idx_left = (i+1)*(N+2) + (j-1+1);
        int idx_right = (i+1)*(N+2) + (j+1+1);

        d_nxt[idx] = -d_prv[idx] + 2.0*d_current[idx]
                   + (dt*dt*c*c)/(dx*dy) * (
                      d_current[idx_up] + d_current[idx_down] + 
                      d_current[idx_left] + d_current[idx_right] - 
                      4.0*d_current[idx]
                   );
    }
    cg::this_grid().sync(); 
}
// END: T5


// TASK: T7
// Main time integration.
void simulate( void )
{
// BEGIN: T7
    // Calculate the dimensions of the grid of blocks.
    dim3 block(BLOCKX, BLOCKY);
    // Even if XSIZE or YSIZE is not evenly divisible by BLOCKX or BLOCKY it will divide it as evenly as possible.
    dim3 grid((M + BLOCKX - 1) / BLOCKX, 
             ( N + BLOCKY - 1) / BLOCKY);

    // Debug prints
    printf("Grid dimensions: %dx%d\n", grid.x, grid.y);
    printf("Block dimensions: %dx%d\n", block.x, block.y);
    printf("Domain dimensions: %ldx%ld\n", M, N);
    printf("dt=%f, dx=%f, dy=%f, c=%f\n", dt, dx, dy, c);

    // Go through each time step
    for ( int_t iteration=0; iteration<=max_iteration; iteration++ ) {
        if ( (iteration % snapshot_freq)==0 )
        {
            // Make sure all kernels have completed before copying data
            hipDeviceSynchronize();
            // Copy current state back to host for saving
            hipMemcpy(buffers[1], d_current, (M + 2) * (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
            // Save the current state
            domain_save ( iteration / snapshot_freq );
                    
            // Rotate the time step buffers
            move_buffer_window();
        }
        
        device_boundary_condition<<<grid, block>>>(d_current, N, M);
        // Synchronize the findings
        hipDeviceSynchronize();

        // Compute next time step
        device_time_step<<<grid, block>>>(d_prv, d_current, d_nxt, N, M, dt, dx, dy, c);
        // Synchronize the findings
        hipDeviceSynchronize();
    }
// END: T7
}


// TASK: T8
// GPU occupancy
void occupancy( void )
{
// BEGIN: T8
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &maxActiveBlocks,
        device_time_step,
        BLOCKX * BLOCKY,  // threads per block
        0     // shared memory size
    );
    int activeWarps = maxActiveBlocks * (BLOCKX * BLOCKY) / BLOCKX;  // 32 threads per warp
    int maxWarps = prop.maxThreadsPerMultiProcessor / BLOCKX;

    float occupancyRate = (float) activeWarps / maxWarps;
    printf("Theoretical occupancy: %.2f%%\n", occupancyRate * 100);
// END: T8
}


// TASK: T2
// Make sure at least one CUDA-capable device exists
static bool init_cuda()
{
// BEGIN: T2
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        printf("Error: No CUDA-compatible GPU device found!\n");
        return false;
    }

    // Use first available device
    hipSetDevice(0);
    
    // Get and print device properties
    hipGetDeviceProperties(&prop, 0);
    
    printf("Device name: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimensions: (%d, %d, %d)\n", 
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
    
    return true;
// END: T2
}


// TASK: T3
// Set up our three buffers, and fill two with an initial perturbation
void domain_initialize ( void )
{
// BEGIN: T3
    // Check if CUDA exists
    if (!init_cuda())
    {
        fprintf(stderr, "CUDA initialization failed\n");
        exit( EXIT_FAILURE );
    }
    // Calculate the necessary memory that must be allocated
    size_t size = (M + 2) * (N + 2) * sizeof(real_t);
    
    // Initialize ALL host buffers using calloc
    buffers[0] = (real_t*)calloc((M + 2) * (N + 2), sizeof(real_t));
    buffers[1] = (real_t*)calloc((M + 2) * (N + 2), sizeof(real_t));
    buffers[2] = (real_t*)calloc((M + 2) * (N + 2), sizeof(real_t));
  
    // Allocate 3 buffers to the device
    hipMalloc((void**) &d_prv, size);
    hipMalloc((void**) &d_current, size);
    hipMalloc((void**) &d_nxt, size);

    for ( int_t i=0; i<M; i++ )
    {
        for ( int_t j=0; j<N; j++ )
        {
            // Calculate delta (radial distance) adjusted for M x N grid
            real_t delta = sqrt ( ((i - M/2.0) * (i - M/2.0)) / (real_t)M +
                                  ((j - N/2.0) * (j - N/2.0)) / (real_t)N );
            U_prv(i,j) = U(i,j) = exp ( -4.0*delta*delta );
        }
    }
    // Copy initialized onto device
    hipMemcpy(d_prv, buffers[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_current, buffers[1], size, hipMemcpyHostToDevice);
    hipMemcpy(d_nxt, buffers[2], size, hipMemcpyHostToDevice);

    // Set the time step for 2D case
    dt = dx*dy / (c * sqrt (dx*dx+dy*dy));
// END: T3
}


int main ( void )
{
    // Set up the initial state of the domain
    domain_initialize();

    struct timeval t_start, t_end;

    gettimeofday ( &t_start, NULL );
    simulate();
    gettimeofday ( &t_end, NULL );

    printf ( "Total elapsed time: %lf seconds\n",
        WALLTIME(t_end) - WALLTIME(t_start)
    );

    occupancy();

    // Clean up and shut down
    domain_finalize();
    exit ( EXIT_SUCCESS );
}
